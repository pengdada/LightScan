#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
//#include ""
#include "cudaLib.cuh"
#include <stdio.h>

template<typename T>
__device__ __forceinline__ T ldg(const T* ptr) {
#if __CUDA_ARCH__ >= 350
    return __ldg(ptr);
#else
    return *ptr;
#endif
}

__forceinline__ __device__ unsigned int getLaneid() {
	unsigned int laneId;
	asm volatile("mov.u32 %0, %laneid;" : "=r"(laneId));
	return laneId;
}

__forceinline__ __device__ unsigned int getWarpid() {
	return threadIdx.x >> 5;
}



template<typename T>
__global__ void scanLF(const T *input, T*output, int n)
{
	auto x = blockDim;
	auto y = gridDim;

	unsigned int warpId, laneId;
	asm volatile("mov.u32 %0, %laneid;" : "=r"(laneId));
	warpId = threadIdx.x >> 5;
	assert(laneId == threadIdx.x % 32);

	T a, elem;

	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	elem = input[tid];

	#pragma unroll
	for (int i = 1; i <= 32; i <<= 1) {
		#pragma unroll
		for (int j = 1; j <= i; j++) {
			a = __shfl_up(elem, j);
			if ((laneId % (i<<1)) == (i-1+j)) {
				elem += a;
			}
		}
	}
	__shared__ int sMem[1024];
	sMem[tid] = elem;
	__syncthreads();

	a = elem;

	//for (int i = 1; i <= 32; i <<= 1) {
	//	/*the first row of the matrix*/
	//	val = __shfl_up(elem[s], i);
	//	T va = val;
	//	if (laneId >= i) {
	//		elem[s] = op(elem[s], val);
	//	}
	//}


	//__shared__ T temp[1024 * 2];
	//int tdx = threadIdx.x; int offset = 1;
	//temp[2 * tdx] = input[2 * tdx];
	//temp[2 * tdx + 1] = input[2 * tdx + 1];

	//for (int d = n >> 1; d > 0; d >>= 1)
	//{
	//	__syncthreads();
	//	if (tdx < d)
	//	{
	//		int ai = offset*(2 * tdx + 1) - 1;
	//		int bi = offset*(2 * tdx + 2) - 1;
	//		temp[bi] += temp[ai];
	//	}
	//	offset *= 2;
	//}
	//if (tdx == 0) temp[n - 1] = 0;
	//for (int d = 1; d < n; d *= 2)
	//{
	//	offset >>= 1; __syncthreads();
	//	if (tdx < d)
	//	{
	//		int ai = offset*(2 * tdx + 1) - 1;
	//		int bi = offset*(2 * tdx + 2) - 1;
	//		float t = temp[ai];
	//		temp[ai] = temp[bi];
	//		temp[bi] += t;
	//	}
	//}
	//__syncthreads();

	//output[2 * tdx] = temp[2 * tdx];
	//output[2 * tdx + 1] = temp[2 * tdx + 1];

}

int mainLF(int argc, char** argv) {
	int SIZE = 2048;
	std::vector<int> vecIn(SIZE), vecOut(SIZE);

	for (int i = 0; i < SIZE; i++) {
		//vecIn[i] = i + 1;
		vecIn[i] = 1;
	}

	DevData<int> devIn(SIZE), devOut(SIZE);
	devIn.CopyFromHost(&vecIn[0], vecIn.size(), vecIn.size(), 1);
	devOut.Zero();
	dim3 grids(1, 1, 1), blocks(SIZE / 2, 1, 1);
	scanLF<< <grids, blocks >> > (devIn.GetData(), devOut.GetData(), SIZE);
	devOut.CopyToHost(&vecOut[0], vecOut.size(), vecOut.size(), 1);
	hipDeviceSynchronize();

	//devOut.CopyToHost(&vecOut[0], 1, 1, 1);


	return 0;
}

